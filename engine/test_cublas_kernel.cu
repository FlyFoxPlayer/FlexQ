#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(status)                                   \
    do {                                                      \
        hipError_t err = status;                             \
        if (err != hipSuccess) {                             \
            std::cerr << "CUDA error: " << hipGetErrorString(err) \
                      << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE);                                \
        }                                                     \
    } while (0)

#define CHECK_CUBLAS(status)                                  \
    do {                                                      \
        hipblasStatus_t err = status;                          \
        if (err != HIPBLAS_STATUS_SUCCESS) {                   \
            std::cerr << "cuBLAS error: "                     \
                      << cublasGetStatusString(err)           \
                      << " at line " << __LINE__ << std::endl;\
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

#define NUM_PROFILE 1000

// CPU implementation of int8 matrix multiplication
void cpu_int8_gemm(int8_t* A, int8_t* B, int32_t* C, 
                   int m, int n, int k) {
    // Clear output matrix
    for (int i = 0; i < m * n; i++) {
        C[i] = 0;
    }
    
    // Matrix multiplication: C = A * B
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int32_t sum = 0;
            for (int l = 0; l < k; l++) {
                // A is row-major (m×k), B is column-major (k×n)
                sum += static_cast<int32_t>(A[i * k + l]) * 
                       static_cast<int32_t>(B[l * n + j]);
            }
            C[i * n + j] = sum;
        }
    }
}

// Function to verify GPU results using CPU calculation
bool verify_results(int32_t* cpu_c, int32_t* gpu_c, int size) {
    for (int i = 0; i < size; i++) {
        if (cpu_c[i] != gpu_c[i]) {
            std::cerr << "Mismatch at index " << i 
                      << ": CPU=" << cpu_c[i] 
                      << ", GPU=" << gpu_c[i] 
                      << std::endl;
            return false;
        }
    }
    return true;
}

void run_cublas_gemm(int m, int n, int k) {
    // Set matrix dimensions and sizes
    size_t size_A = m * k * sizeof(int8_t);
    size_t size_B = k * n * sizeof(int8_t);
    size_t size_C = m * n * sizeof(int32_t);
    uint64_t seed = 0x2019;

    // Allocate host memory using malloc
    int8_t* h_A = (int8_t*)malloc(size_A);
    int8_t* h_B = (int8_t*)malloc(size_B);
    int32_t* h_C = (int32_t*)malloc(size_C);
    int32_t* h_cpu_C = (int32_t*)malloc(size_C);
    int32_t* h_gpu_C = (int32_t*)malloc(size_C);

    // Initialize host matrices
    srand(seed);
    for (int i = 0; i < m * k; i++) 
        h_A[i] = static_cast<int8_t>(rand() % 256 - 128); // [-128, 127]
    for (int i = 0; i < k * n; i++) 
        h_B[i] = static_cast<int8_t>(rand() % 256 - 128); // [-128, 127]

    // // Compute CPU reference result
    // cpu_int8_gemm(h_A, h_B, h_cpu_C, m, n, k);

    // Allocate device memory
    int8_t *d_A, *d_B;
    int32_t *d_C;
    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set gemm parameters
    const int32_t alpha = 1;
    const int32_t beta = 0;
    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;

    // cuBLAS dimensions:
    int lda = k; // leading dimension for A 
    int ldb = n; // leading dimension for B
    int ldc = n; // leading dimension for C/D
    
    // Timing measurement
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_PROFILE; ++i) {
        CHECK_CUBLAS(hipblasGemmEx(
            handle, transA, transB,
            n, m, k,
            &alpha,
            d_B, HIP_R_8I, ldb,
            d_A, HIP_R_8I, lda,
            &beta,
            d_C, HIP_R_32I, ldc,
            HIPBLAS_COMPUTE_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP
        ));
    }

    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // // Copy GPU result back to host
    // CHECK_CUDA(cudaMemcpy(h_gpu_C, d_C, size_C, cudaMemcpyDeviceToHost));

    // // Verify GPU result against CPU result
    // if (!verify_results(h_cpu_C, h_gpu_C, m * n)){
    //     printf("Results mismatch!");
    //     exit(EXIT_FAILURE);
    // }

    // Calculate performance metrics
    double avg_time = milliseconds / NUM_PROFILE;
    double flops = 2.0 * m * n * k;
    double tflops = (flops * 1e-12) / (avg_time / 1000.0);

    printf("cuBLAS-W8A8-GEMM. m: %6d, n: %6d, k: %6d,\t Time: %.4f ms, TFLOPS: %4.4f\n",
           m, n, k, avg_time, tflops);

    // Clean up resources
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_cpu_C);
    free(h_gpu_C);
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s M N K\n", argv[0]);
        printf("Example: %s 1 4096 4096\n", argv[0]);
        return -1;
    }

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);

    // Verify GPU compute capability
    hipDeviceProp_t props;
    CHECK_CUDA(hipGetDeviceProperties(&props, 0));
    
    if (props.major < 7) {
        std::cerr << "cuBLAS int8 GEMM requires compute capability 70+ (Volta+)\n";
        return 0;
    }

    run_cublas_gemm(m, n, k);
    return 0;
}